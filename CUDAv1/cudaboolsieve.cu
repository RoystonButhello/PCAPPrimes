#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

typedef unsigned __int64 integer;
const size_t size = sizeof(integer);
const integer pow2_32 = 4294967296;
const int threads = 8;
__constant__ integer dpow2_32 = pow2_32;
__constant__ int dthreads = threads;
__constant__ integer segsize = pow2_32 / threads;

//Utility function to calculate postive integer-powers
integer power(integer val, integer exp)
{
	integer temp = val;
	for (integer i = 1; i < exp; i++) temp *= val;
	return temp;
}

//Utility function to approximate no. of primes between 1->n as n/ln(n)
integer trimSize(integer n)
{
	long double e = 2.7183;
	integer exp = 1;
	while (pow(e, exp) < n)
		exp++;
	return n / (exp - 2);
}

///////////////////////////KERNEL START///////////////////////////
__global__ void SieveBlock(integer *P, bool *mark, integer completed)
{
	//Each thread sieves [(pow2_32 >> 1) / threads] elements of the current block
	__shared__ integer global_min;
	global_min = completed * dpow2_32 + 1;
	integer id = threadIdx.x, i, j, minb;
	integer min = global_min + (id * segsize);
	integer max = min + segsize - 2;

	for (i = 0;P[i] * P[i] <= max;i++)
	{
		minb = (min / P[i]) * (P[i]) + P[i];
		if (~minb & 1)	minb += P[i];
		for (j = minb;j <= max;j += (P[i] << 1))
			mark[(j - (global_min - 1)) >> 1] = true;
	}
	printf("Kernel %3llu stopped at %llu [%11llu -> %11llu]\n", id + completed, j - (P[i] << 2), min, max);
	/*for (j = max; j >= min; j -= 2)
	{
		if (!mark[(j - min + 1) >> 1])
		{
			printf("Kernel %llu: %llu\n", id , j);
			break;
		}
	}*/
}
////////////////////////////KERNEL END////////////////////////////

//     SEGMENTED SIEVE
//	n		RAM	    Time
// E07	   552KB   0.026s
// E08	   620KB   0.206s
// E09	   704KB   1.895s
// E10	   668KB   20.02s 
// E11     904KB   205.2s

//PARALLEL SEGMENTED SIEVE
//	n		RAM	    Time
// E07	   203MB   0.481s
// E08	   202MB   4.405s 
// E09	      
// E10	      
// E11      

//Stats logged via Visual Studio Performance Profiler on i7 4790K @4.00GHz w/ 16GB DDR3 RAM and GTX 1070Ti
//Can't take n>

//Driver function
int main(int argc, char* argv[])
{
	//Range: Data-type dependent
	integer n;
	printf("Enter n: ");
	scanf("%llu", &n);

	integer m = sqrt(n);
	integer marklen = n >> 1;

	bool smallsieve = false;	//Use serial sieve for n<2^32
	if (n <= pow2_32)
		smallsieve = true;
	else if (n % pow2_32 > 0)	//If n>2^32 then round n to nearest multiple of 2^32
	{
		printf("Rounded %llu to ", n);
		n = ((n / pow2_32) + 1) * pow2_32;
		printf("%llu\n\n", n);
		m = 65536;				//sqrt(pow2_32)
		marklen = pow2_32 >> 1;
	}

	integer limit = (smallsieve) ? n : pow2_32;

	integer plen = trimSize(pow2_32);
	if (~n & 1) n--;
	if (~m & 1) m--;

	//Boolean array initialized to false
	bool *mark = (bool *)calloc(marklen + 1, sizeof(bool));	//Represents [2,3,5,7,9,11,...,sqrt(n)]

	//Array to store primes b/w [2,m]
	integer *P = (integer *)calloc(plen + 1, (size_t)size);
	if (mark == NULL || P == NULL) { printf("Memory Allocation Failed!\n"); exit(1); }
	integer i, j, k, offset;

	//Log execution time
	clock_t START_TIME, END_TIME;
	double  CPU_TIME = 0.0;
	float GPU_TIME = 0.0;
	float temp_t;

	//Setup-Phase: Calculate all primes in the range [3,m]
	START_TIME = clock();
	for (i = 5, k = 1, offset = 2; i < m; i += offset, offset = 6 - offset)	//i->[3,5,7,9...,sqrt(n)] | i corresponds to mark[(i-3)/2]
	{
		if (!mark[i >> 1])
		{
			if (i*i <= limit)
				for (j = i * i; j <= limit; j += (i << 1))	//j->[i^2,n] | increments by 2*i
					mark[j >> 1] = 1;
			P[k++] = i;
		}
	}
	END_TIME = clock();
	CPU_TIME = ((double)(END_TIME - START_TIME)) / CLOCKS_PER_SEC;

	printf("Stopped primary sieve at prime %llu\n", P[k - 1]);
	for (;i <= limit;i += offset, offset = 6 - offset)
	{
		if (!mark[i >> 1])
			P[k++] = i;
	}

	P[0] = 3;
	plen = k;
	free(mark);
	printf("Last prime: %llu @ index [%llu]\n\n", P[plen - 1], plen - 1);
	if (smallsieve)
	{
		free(P);
		printf("\nSETUP-PHASE CPU Time: %0.3f seconds\n", CPU_TIME);
		return 0;
	}

	integer chunksize = pow2_32 >> 1;						//Elements per chunk of 2^32 digits
	integer chunkcount = (n - pow2_32 - 1) / chunksize;		//No. of chunks
	integer completed = 1;
	printf("%llu chunk(s) for [%llu->%llu]\n", chunkcount, pow2_32 - 1, n);

	integer *dP;
	bool *dmark;

	//CUDA Malloc
	hipMalloc(&dP, (plen + 1) * (size));
	hipMalloc(&dmark, chunksize);

	//Calculate dimensions
	dim3 TPB(threads, 1, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(dP, P, plen * size, hipMemcpyHostToDevice);

	while (completed <= chunkcount)
	{
		mark = (bool *)calloc(chunksize + 1, sizeof(bool));
		hipMemcpy(dmark, mark, chunksize, hipMemcpyHostToDevice);

		hipEventRecord(start);
		SieveBlock << <1, TPB >> > (dP, dmark, completed);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&temp_t, start, stop);

		GPU_TIME += temp_t;
		completed++;
		free(mark);
	}

	free(P);
	hipFree(dP);

	GPU_TIME /= 1000;
	printf("\nSETUP-PHASE CPU Time: %0.3f seconds\n", CPU_TIME);
	printf("COMPUTE-PHASE GPU Time: %0.3f seconds\n", GPU_TIME);
	return 0;

}

